#include "hip/hip_runtime.h"


#include <iostream>

static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    std::cerr << file << " " << line << " " << hipGetErrorString(err) << std::endl;
    std::exit(-1);
  }
}

#define HANDLE_ERROR(err) HandleError(err, __FILE__, __LINE__)

// __global__修饰符告诉编译器，该函数在设备上运行
__global__ void add(const int* a, const int* b, int* c, int size) {
  // blockIdx.x Block的索引
  int block_index = blockIdx.x + blockIdx.y * gridDim.x;
  int index = block_index * blockDim.x + threadIdx.x;
  if (index < size)
    c[index] = b[index] + a[index];
  // __shared__ 标识共享内存，由一个线程块内的多个线程共享
  __shared__ float cache[32];
  int cache_index = threadIdx.x;
  cache[cache_index] = cache_index;
  // 对线程块中的线程进行同步，确保线程块中每个线程都执行完之前的语句后才执行下面的语句
  __syncthreads();
  // 由每个线程块的第一个线程执行规约操作
  if (threadIdx.x == 0) {
    int sum = 0;
    for(int i = 0; i < 32; ++i) {
      sum = sum + cache[i];
    }
    cache[0] = sum;
    c[block_index] = cache[0];
  }
}

extern "C" {
  void addKernel(int* a, int* b, int* c, int size) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipSetDevice(0);
    // cudaMalloc在设备上分配内存，不能在主机代码中使用cudaMalloc分配的指针进行内存读写操作
    hipMalloc((void**)&dev_a, sizeof(int) * size);
    hipMalloc((void**)&dev_b, sizeof(int) * size);
    hipMalloc((void**)&dev_c, sizeof(int) * size);
    hipMemcpy(dev_a, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, sizeof(int) * size, hipMemcpyHostToDevice);
    // 第一个参数指定执行核函数的并行线程块的数量(Block)，当使用数字时，默认为一维；使用dim3类型的变量指定2/3维数据
    // 第二个参数指定每个线程块中线程的数量
    // 这两个值都可以是多维的
    // 线程块的数量上限为65535
    // 每个线程块中线程的数量限制为maxThreadsPerBlock
    // 这些Block的集合也称为一个Grid
    // 线程块为2维度，共9个，每个块中有32个线程
    dim3 grid(3, 3);
    add<<<grid, 32>>>(dev_a, dev_b, dev_c, size);
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << " use time : ms" << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
}

int main() {
  // 获取cuda设备的数量
  int count = 0; 
  HANDLE_ERROR(hipGetDeviceCount(&count));
  std::cout << "cuda device's count " << count << std::endl;

  hipDeviceProp_t prop;
  for(int i = 0; i < count; ++i) {
    // 获取指定cuda设备的属性
    HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
    std::cout << "name : " << prop.name << std::endl;
    std::cout << "global mem(GB) : " << prop.totalGlobalMem / ( 1024 * 1024 * 1024.0) << std::endl;
  }

  int a[20];
  int b[20];
  int c[20];

  for(int i = 0; i < 20; ++i) {
    a[i] = i;
    b[i] = 2 * i;
    c[i] = 0;
  }

  addKernel(a, b, c, 20);
  for(int i = 0; i < 20; ++i) {
    std::cout << c[i] << " " ;
  }
  return 0;
}